#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <time.h>
#include <thread>

__global__ void add(float* x, float* y, float* z, int n)
{
    // ��ȡȫ������
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // ����
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] * y[i];
    }
}

/*
int main(void)
{
    int N = 1e8;
    int nBytes = N * sizeof(float);

    // �����й��ڴ�
    float* x, * y, * z1, *z2;
    cudaMallocManaged((void**)&x, nBytes);
    cudaMallocManaged((void**)&y, nBytes);
    cudaMallocManaged((void**)&z1, nBytes);

    // ��ʼ������
    for (int i = 0; i < N; ++i)
    {
        x[i] = 11110.0;
        y[i] = 22220.0;
    }

    clock_t start1 = clock();
    for (int i = 0; i < N; i++) {
        z2[i] = x[i] * y[i];
    }
    clock_t end1 = clock();
    clock_t start2 = clock();
    // ����kernel��ִ������
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    // ִ��kernel
    add << < gridSize, blockSize >> > (x, y, z1, N);
    // ͬ��device ��֤�������ȷ����
    cudaDeviceSynchronize();
    // ���ִ�н��
    clock_t end2 = clock();
    
    // output
    double t1 = ((double)(end1 - start1)) / CLOCKS_PER_SEC;
    double t2 = ((double)(end2 - start2)) / CLOCKS_PER_SEC;
    std::cout << "time (CPU): " << t1 * 1000 << " ms" << std::endl;
    std::cout << "time (GPU): " << t2 * 1000 << " ms" << std::endl;

    // �ͷ��ڴ�
    cudaFree(x);
    cudaFree(y);
    cudaFree(z1);
    cudaFree(z2);

    return 0;
}
*/